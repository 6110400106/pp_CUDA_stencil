
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1000000
#define R 3
#define BLOCK_SIZE 1024

__global__ void singlethread_stencil(int* d_in, int* d_out, int M) {
	int tid = threadIdx.x;
	if(tid == 0) {
		for(int i = 0; i < M; i++) {
			for(int j = -R; j <= R; j++) {
				d_out[i] += d_in[i+j+R];
			}
		}
	}
}

__global__ void multiplethreads_stencil(int* d_in, int* d_out, int M) {
	int tid = (blockIdx.x * BLOCK_SIZE) + threadIdx.x;
	if(tid < M) {
		int result = 0;
		for(int j = -R; j <= R; j++) {
			result += d_in[tid+R+j];
		}
		d_out[tid] = result;
	}
}

__global__ void faster_stencil(int* d_in, int* d_out, int M) {
	__shared__ int temp[BLOCK_SIZE+(2*R)];

	int g_id =  (blockIdx.x * BLOCK_SIZE) + threadIdx.x;
	int l_id = threadIdx.x + R;

	if(g_id < M) {
		
		temp[l_id] = d_in[g_id+R];

		if(threadIdx.x < R) {
			temp[l_id-R] = d_in[g_id];
			temp[l_id+BLOCK_SIZE] = d_in[g_id+BLOCK_SIZE];
		}

		__syncthreads();

		int result = 0;

		for(int j = -R; j <= R; j++) {
			result += temp[l_id+R];
		}
		d_out[g_id] = result;
	}

}

int main() {
	int M = N-2*R;
	int h_in[N];
	int h_out[M];
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	for(int i = 0; i < N; i++) {
		h_in[i] = 1;
	}
	
	int* d_in;
	int* d_out;
	
	//Part 1: Memory transfer from host to device
	hipMalloc((void**) &d_in, N*sizeof(int));
	hipMalloc((void**) &d_out, M*sizeof(int));

	hipMemcpy(d_in, &h_in, N*sizeof(int), hipMemcpyHostToDevice);
	
	//Part 2: Execute kernel
	
	//Timed singlethread_stencil time
	/*cudaEventRecord(start);
	singlethread_stencil<<<1, BLOCK_SIZE>>>(d_in, d_out, M);
	cudaEventRecord(stop);*/

	//Timed multiplethreads_stencil time
	/*cudaEventRecord(start);
	multiplethreads_stencil<<<ceil(M/(double) BLOCK_SIZE), BLOCK_SIZE>>>(d_in, d_out, M);
	cudaEventRecord(stop);*/
	
	//Timed faster_stencil time
	hipEventRecord(start);
	faster_stencil<<<ceil(M/(double) BLOCK_SIZE), BLOCK_SIZE>>>(d_in, d_out, M);
	hipEventRecord(stop);

	//Part 3: Memory transfer from device to host
	hipMemcpy(&h_out, d_out, M*sizeof(int), hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	hipFree(d_in);
	hipFree(d_out);

	//Part 4: Check the result
	
	for(int i = 0; i < M; i++) {
		if(h_out[i] != 2*R+1) {
			printf("Incorrect result!\n");
			return -1;
		}
		//printf("%d ", h_out[i]);
	}
	printf("Correct result!\n");
	printf("Time used: %f milliseconds\n", milliseconds);

	return -1;


}
